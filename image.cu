#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctype.h>
#include "getopt.h"
#include <stdint.h>
#include <errno.h>
#include <iostream>

using namespace std;

// CURAND library
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// NPP Library
#include <ImagesNPP.h>
#include <ImagesCPU.h>
#include <npp.h>
#include <nppdefs.h>
#include <nppi_arithmetic_and_logical_operations.h>

// Local includes
#include "image.hpp"   // Library
#include "main.hpp"    // Standard functionality interfaces
#include "cxxopts.hpp" // Command Line Argument Parsing (TODO: Move into library)

__constant__ uint32_t chromakey = 0x11111111;
/* NOTE: while above variable is defined in host namespace, it's value
 *  is not accessible. Trying to access it from the CPU directly will
 *  not yield any errors or warnings, but will always read with a 0-value.
 * The value can be read using hipMemcpyFromSymbol(), but in most cases it's
 *   simpler to store a second copy in host memory for convenience.
 */

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}



// Pipe two data set values together (bitwise-or)
__global__
void data_merge(unsigned int * data, unsigned int * data2)
{
  // blockNum * thradsPerBlock + threadNum
  const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  data[thread_idx] = data[thread_idx] | data2[thread_idx];
	
}

// Adjust data set by bitwise oring all values against a constant value (chromakey)
__global__
void key_merge(uint32_t *data, uint32_t const opt)
{
  const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (opt == 0) {
    data[thread_idx] = (data[thread_idx] | chromakey);
  } else {
    data[thread_idx] = (data[thread_idx] & chromakey);
  }
}

/** Flip the image horizontally
 *   This basic version assumes that the number of threads is equal to the
 *      widtdh of the image, and blocks to the height.
 */ 
__global__
void flip_image_row(uint32_t *data)
{
  extern __shared__ int row[];
  const int x = threadIdx.x; // col
  const int nx = blockDim.x; // num_threads = number of columns
  const int ybase = blockIdx.x * nx; // start of row
  
  row[x] = data[ybase+x];
  __syncthreads();
  data[ybase + nx-1 - x] = row[x];
}

/** Flip the image vertically
 *  This basic version assumes that the number of threads is equal to the
 *    height of the image, and blocks to the width.
 */ 
__global__
void flip_image_col(uint32_t *data)
{
  extern __shared__ int col[];
  const int y = threadIdx.x;
  const int dim = blockDim.x;
  const int x = blockIdx.x;

  const unsigned int idx = (y * dim) + x;
  col[y] = data[idx];
  __syncthreads();
  data[idx] = col[dim - 1 - y];
    
}

/**
 *   add random noise to the image with per-channel noise bound
 *    by the defined max noise levels (in chromakey).
 */
__global__
void add_noise(uint32_t *data, unsigned int seed)
{
  const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int noise;
  char ch_noise;

  hiprandState_t state;
  hiprand_init(thread_idx*seed,0,0,&state);
  
  noise = hiprand(&state);

  ch_noise = GET_R(noise) % GET_R(chromakey);
  noise = SET_R(noise,ch_noise);
  ch_noise = GET_G(noise) % GET_G(chromakey);
  noise = SET_G(noise,ch_noise);
  ch_noise = GET_B(noise) % GET_B(chromakey);
  noise = SET_B(noise,ch_noise);  

  data[thread_idx] += noise;
}

/**
 * This kernel should be executed once for every pixel
 */
__global__
void gpu_steg_image_en(uint32_t *data, char *msg, uint32_t msg_len)
{
  const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int tmp;
  unsigned char halfByte;
  
  if (thread_idx < msg_len*2)
  {
    tmp = data[thread_idx];

    // Load the half-byte from source message
    if (threadIdx.x & 0x1 == 1) {
      // Odd threads take the upper half-byte
      halfByte = (msg[(thread_idx-1)/2]) >> 4;
    } else {
      // Even threads take the lower half-byte
      halfByte = msg[thread_idx/2];
    }

    // Add cipher (Note: We don't need to mask overflow bytes, since we select bits below)
    halfByte += chromakey;

    // Bit-1 of char to Bit-1 of R
    tmp = tmp ^ (halfByte & 0x1);

    // Bit-2 of char to Bit-1 of G
    tmp = tmp ^ ((halfByte & 0x2) << 7);

    // Bit-3+4 of char to Bit-1+2 of B
    tmp = tmp ^ ((halfByte & 0xC) << 14);

      
    data[thread_idx] = tmp;
  } else {
    // Nothing to be done
  }
  
}

/**
 *  This kernel should be executed once per pixel
 */
__global__
void gpu_steg_image_de(uint32_t *data, uint32_t *data2, char *msg_out)
{
  extern __shared__ char msg[];
  const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  uint32_t tmp, tmp2;

  // Difference the two images using XOR
  tmp = data2[thread_idx] ^ data[thread_idx];

  // Calculate the half-word
  tmp2 = GET_R(tmp) & 0x1; // Bit 1
  tmp2 |= (GET_G(tmp) & 0x1) << 1; // Bit 2
  tmp2 |= (GET_B(tmp) & 0x3) << 2; // Bits 3+4
  msg[threadIdx.x] = tmp2;

  // Sync threads
  __syncthreads();

  /* Note: This next part could be somewhat optimized in theory if we
     could keep our shared memory but switch to a kernel with half the
     block size.
   */
  if (threadIdx.x & 0x1 == 1) {
    // Only odd threads will proceed

    // Merge the half-words and apply the cipher (to each half)
    tmp =   ((int)(msg[threadIdx.x-1]) - chromakey) & 0xF;
    tmp |= (((int)(msg[threadIdx.x]) - chromakey) & 0xF) << 4;

    // Output decrypted character
    msg_out[ (thread_idx-1)/2 ] = tmp;
    
  } else {
    // Even threads are now idle/stalled
  }
  
}


/** Overlay sprite onto image starting at given x offset
 *   This version will generate num_frames images, waiting on
 *   an event in between executions.
 *    Note: For proper operation, num_threads=width and numBlocks=height of
 *      src image.
 *   Pixels corresponding to the chromakey will not be copied over.
 */
__global__
void gpu_img_sprite(unsigned int* src, unsigned int* sprite,
		    unsigned int sprite_width, unsigned int sprite_height,
		    unsigned int* gpu_out,
		    unsigned int const sprite_offset
		    )
{
  const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int sprite_x, sprite_y;
  unsigned int sprite_idx;
  
  sprite_x = threadIdx.x - sprite_offset;
  sprite_y = blockIdx.x - sprite_offset;
  sprite_idx = (sprite_y * sprite_height) + sprite_x;

  if (sprite_x < 0 || sprite_y < 0 || sprite_y > sprite_width || sprite_x > sprite_height || sprite[sprite_idx] == chromakey) {
    gpu_out[thread_idx] = src[thread_idx];
  } else {
    gpu_out[thread_idx] = sprite[sprite_idx];
  }
}

/** Simple (Naive) Convolution Example 
 * Based on https://github.com/bgaster/opencl-book-samples/blob/master/src/Chapter_3/OpenCLConvolution/Convolution.cl
 *
 * This simple implementation is not optimized (and therefore more readable).
 *
 * This simple implementation requires image and mask to each be square. Image must be small enough to run
 *   one image row per GPU block.
 * TODO: If requirement remains in place, enforce it 
**/
__global__ void gpu_convolve(
                         unsigned int * const input,
                         int * const mask,
                         unsigned int * const output,
                         const int inputWidth,
                         const int maskWidth)
{
  const int x = threadIdx.x; //get_global_id(0);
  const int y = blockIdx.x; //get_global_id(1);

  uint sum = 0;
  for (int r = 0; r < maskWidth; r++)
    {
      const int idxIntmp = (y + r) * inputWidth + x;

      for (int c = 0; c < maskWidth; c++)
        {

          sum += mask[(r * maskWidth)  + c] * input[idxIntmp + c];
        }
    }

  output[y * inputWidth + x] = sum;
}


/*** OpenCL Application ***/
class GIMD_cuda : virtual public GIMD_main
{
  hipDeviceProp_t deviceProp;
  uint32_t num_blocks, num_threads;
  bool use_npp = false;
  
public:
  /** Constructor verifies device requirements and initializes hardware references */
  GIMD_cuda()
  {
    // Get and output basic device information
    if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0)) {
      throw std::runtime_error("ERROR: Unable to get device properties.\n");
    } else {
      printf("INFO: GPU supports a warpSize of %d, and a maximum of %d threads per block\n",
             deviceProp.warpSize,
             deviceProp.maxThreadsPerBlock
             );
    }
  }
  
  int run(int argc, char* argv[])
  {
    string msg = NULL;

    // Parse Arguments
    cxxopts::Options options = init_options("OpenCL Test App");
    options.add_options()
      ("b,blocks", "Number of blocks", cxxopts::value<uint32_t>(num_blocks))
      ("t,threads", "Number of threads", cxxopts::value<uint32_t>(num_threads))
      ("p,pinned", "Use PINNED memory where applicable")
      ("npp", "Use NPP Library for supported functions", cxxopts::value<bool>(use_npp))
      ;
    cxxopts::ParseResult result = parse_options(options, argc, argv);
    cout << "Arguments parsed" << endl;

    // Set chromakey constant for CUDA
    hipMemcpyToSymbol(HIP_SYMBOL(chromakey), &host_chromakey, sizeof(int));

    if (result.count("pinned"))
    {
      memMode = MEM_HOST_PINNED;
    }

    // Execute action
    do_action();  
	
    return 0;
  }
  
  int mod_image(uint32_t* data, uint32_t **output, const uint32_t height, const uint32_t width )
  {
    if (use_npp) {
      return npp_mod_image(data, height, width);
    } else {
      return cuda_mod_image(data, height, width);
    }
  }

  int cuda_mod_image(uint32_t* data, const uint32_t height, const uint32_t width )
  {
    cuda_img_precheck(height, width);
    
    // Data/Array size defined in this example to match thread/block configuration
    uint32_t data_size = num_threads*num_blocks*sizeof(uint32_t);
    
    /* Declare pointers for GPU based params */
    unsigned int *gpu_data;
    
    // Define performance metrics
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc((void **)&gpu_data, data_size);
    
    hipMemcpy( gpu_data, data, data_size, hipMemcpyHostToDevice );
    
    /* Execute our kernel */
    hipEventRecord(start);
    switch(mode) {
    case MODE_ADD_RAND_NOISE:
      add_noise<<<num_blocks, num_threads>>>(gpu_data, time(NULL));
      break;
    case MODE_OR_MASK:
    case MODE_AND_MASK:
      key_merge<<<num_blocks, num_threads>>>(gpu_data, mode);
      break;
    case MODE_FLIP_HOR:
      // Third parameter dynamically allocates shared memory
      flip_image_row<<<num_blocks, num_threads, num_threads*sizeof(int)>>>(gpu_data);
      break;
    case MODE_FLIP_VER:
      // Third parameter dynamically allocates shared memory
      flip_image_col<<<num_blocks, num_threads, num_blocks*sizeof(int)>>>(gpu_data);
      break;
    default:
      printf("ERROR: Invalid mode (%d) passed to mod_image function\n", mode);
    }
    
    // Wait for the GPU launched work to complete
    //   (failure to do so can have unpredictable results)
    hipDeviceSynchronize();	
    
    hipEventRecord(stop);
    
    /* Free the arrays on the GPU as now we're done with them */
    hipMemcpy( data, gpu_data, data_size, hipMemcpyDeviceToHost );
    hipFree(gpu_data);
    
    /* Iterate through the arrays and output */
    if (verbose) {
      for(unsigned int i = 0; i < num_blocks*num_threads; i++)
        {
          printf("Data: %08x - %03u %03u %03u\n",
                 data[i], GET_R(data[i]), GET_G(data[i]), GET_B(data[i]));
        }
    }
    
    // Output timing metrics
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("mod_image CUDA operation took %f ms\n", milliseconds);
    
    // Report if any errors occurred during CUDA operations
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
      printf("Error: %s\n", hipGetErrorString(err));

    return 1;
  }


  // Simple steganographic message decryption
  int steg_image_de(uint32_t *data, uint32_t **output,
                    const uint32_t image_size, const uint32_t data_height, const uint32_t data_width )
  {
    cuda_img_precheck(data_height, data_width);
    if (num_blocks*num_threads > image_size) {
      throw std::runtime_error("Illegal block/thread size for this image");
    }

    int status;
    uint32_t data2_length=0, height, width;
    uint32_t *data2 = NULL; // Start: Encoded image.  End: Decoded message
    char *msg;
    int msgLen;

    /* Declare pointers for GPU based params */
    unsigned int *gpu_data;
    unsigned int *gpu_data2;
    char *gpu_msg;

    // Define performance metrics
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
  
 
    // Load Encoded Image
    status = load_image(output_file.c_str(), &data2, &data2_length, &height, &width, memMode);
    if (status < 0) {
      printf("ERROR: Unable to load encoded image\n");
      return -1;
    }

    // Validate that lengths match
    if (data2_length != image_size) {
      printf("ERROR: Encoded and source images must be of the same size!\n");
      return -1;
    }

    // Max message length is 1/2 the number of pixels
    msgLen = width*height/2;
    msg = (char*)malloc(msgLen);

    // Load GPU Data
    hipMalloc((void **)&gpu_data, image_size);
    hipMemcpy( gpu_data, data, image_size, hipMemcpyHostToDevice );
    hipMalloc((void **)&gpu_data2, image_size);
    hipMemcpy( gpu_data2, data2, image_size, hipMemcpyHostToDevice );
    hipMalloc((void **)&gpu_msg, msgLen);

    /* Execute our kernel */
    hipEventRecord(start);
    gpu_steg_image_de<<<num_blocks, num_threads, num_threads>>>(gpu_data, gpu_data2, gpu_msg);

    // Wait for the GPU launched work to complete
    //   (failure to do so can have unpredictable results)
    hipDeviceSynchronize();
    hipEventRecord(stop);
  
  
    /* Cleanup */
    hipMemcpy( msg, gpu_msg, msgLen, hipMemcpyDeviceToHost ); 
    hipFree(gpu_data);
    hipFree(gpu_data2);
    hipFree(gpu_msg);
    free(msg);
    switch(memMode) {
    case MEM_HOST_PAGEABLE:
      free(data2);
      break;
    case MEM_HOST_PINNED:
      hipHostFree(data2);
      break;
    }

    printf("Decoded message reads: %s \n\n", msg);
    printf("DEBUG: msg[0]=%x\n", msg[0]);

    // Output timing metrics
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("decryption CUDA operation took %f ms\n", milliseconds);

    // Report if any errors occurred during CUDA operations
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
      printf("Error: %s\n", hipGetErrorString(err));

    *output = data2;
    return 1;
  }

  // Simple steganographic message decryption
  int steg_image_en(uint32_t *data, uint32_t **output, uint32_t image_size)
  {
    if (num_threads == 0) {
      num_threads = 32;
    }
    if (num_blocks == 0) {
      num_blocks = (extra.length() / num_threads)+1;
    }

  
    int msgLen = extra.length();

    /* Declare pointers for GPU based params */
    unsigned int *gpu_data;
    char *gpu_data2;

    // Define performance metrics
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
  
 
    // Load GPU Data
    hipMalloc((void **)&gpu_data, image_size);
    hipMemcpy( gpu_data, data, image_size, hipMemcpyHostToDevice );
    hipMalloc((void **)&gpu_data2, msgLen);
    hipMemcpy( gpu_data2, extra.c_str(), msgLen, hipMemcpyHostToDevice );

    /* Execute our kernel */
    hipEventRecord(start);
    gpu_steg_image_en<<<num_blocks, num_threads, num_threads>>>(gpu_data, gpu_data2, msgLen);

    // Wait for the GPU launched work to complete
    //   (failure to do so can have unpredictable results)
    hipDeviceSynchronize();
    hipEventRecord(stop);
  
    /* Cleanup */
    hipMemcpy( data, gpu_data, image_size, hipMemcpyDeviceToHost ); 
    hipFree(gpu_data);
    hipFree(gpu_data2);

    // Output timing metrics
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("encryption CUDA operation took %f ms\n", milliseconds);

    // Report if any errors occurred during CUDA operations
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
      printf("Error: %s\n", hipGetErrorString(err));

  
    return 1;
}

  int convolve(uint32_t* data, uint32_t **output, const uint32_t height, const uint32_t width )
{
  // Sanity check
  if (data == NULL || !(height == width == num_threads == num_blocks) )
  {
    // TODO: Evaluate this limitation and remove if practical
    throw std::runtime_error("ERROR: This function currently requires height=width=num_threads=num_blocks\n");
  }
  
  // Data/Array size defined in this example to match thread/block configuration
  uint32_t data_size = num_threads*num_blocks*sizeof(uint32_t);
  uint32_t mask_size = sizeof(int32_t)*mask_width*mask_width;
  
  /* Declare pointers for GPU based params */
  unsigned int *gpu_data, *gpu_out;
  int *gpu_mask;

  // Define performance metrics
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipMalloc((void **)&gpu_data, data_size);
  hipMemcpy( gpu_data, data, data_size, hipMemcpyHostToDevice );
  
  hipMalloc((void **)&gpu_out, data_size);
  
  hipMalloc((void **)&gpu_mask, mask_size);
  hipMemcpy( gpu_mask, mask, mask_size, hipMemcpyHostToDevice);

  /* Execute our kernel */
  hipEventRecord(start);
  gpu_convolve<<<num_blocks, num_threads>>>(gpu_data, gpu_mask, gpu_out,
                                            num_threads, // Provided to match original, but not necessary in current config
                                            mask_width);

  // Wait for the GPU launched work to complete
  //   (failure to do so can have unpredictable results)
  hipDeviceSynchronize();	
  
  hipEventRecord(stop);
  
  /* Free the arrays on the GPU as now we're done with them */
  hipMemcpy( data, gpu_out, data_size, hipMemcpyDeviceToHost );
  hipFree(gpu_data);
  hipFree(gpu_out);
  hipFree(gpu_mask);

  /* Iterate through the arrays and output */
  if (verbose) {
    for(unsigned int i = 0; i < num_blocks*num_threads; i++)
      {
	printf("Data: %08x - %03u %03u %03u\n",
	       data[i], GET_R(data[i]), GET_G(data[i]), GET_B(data[i]));
      }
  }

  // Output timing metrics
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("mod_image CUDA operation took %f ms\n", milliseconds);

  // Report if any errors occurred during CUDA operations
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));

  return 1;
}



/* Simple sprite animation.  The given sprite image will be overlaid
 *  on the base image. The first image will start at position 0,0, with
 *  the starting column incremented for each pass.  One image will be generated for each width-1 pixels.
 * The resulting images can be converted into an animated gif using the convert tool:
 *  convert -delay 20 -loop 0 out_fn* out_fn.gif
 *  WARNING: This conversion process can be slow. Ideally, this could be sped up
 *   by utilizing CUDA to directly convert files into GIF format and letting the
 *   CPU assemble the results into an animated GIF ... but one step at a time.
 */
int img_sprite_anim(uint32_t *data, uint32_t **output,
                    uint32_t height, uint32_t width
                    )
{
  // this.extra is sprite filename
  cuda_img_precheck(height, width);
  if (memMode != MEM_HOST_PINNED) {
    throw std::runtime_error("ERROR: Pinned memory required for this operation");
  }
  if (extra.empty()) {
    throw std::runtime_error("Sprite file must be defined for this operation");
  }
    
  uint32_t image_size = width*height;
  char out_fn[64];
  uint32_t num_images = 0;
  uint32_t *sprite_data = NULL;
  uint32_t sprite_length=0, sprite_height, sprite_width;
  uint32_t *gpu_src, *gpu_sprite, *gpu_out1, *gpu_out2;
  uint32_t *cpu_out1, *cpu_out2;
  hipEvent_t start1, stop1, start2, stop2;
  hipStream_t stream1, stream2;
  int status;
  
  // Load sprite image
  status = load_image(extra.c_str(),
		      &sprite_data, &sprite_length, &sprite_height, &sprite_width,
		      MEM_HOST_PINNED);
  if (status < 0) {
    printf("ERROR: Unable to load sprite\n");
    return -1;
  }
  
  // Calculate number of images to generate (must be even to simplify logic)
  num_images = width - (width&1);
  
  // Initialize remaining CUDA resources
  hipHostMalloc((void **)&cpu_out1, image_size);
  hipHostMalloc((void **)&cpu_out2, image_size);

  hipMalloc((void **)&gpu_sprite, image_size);
  hipMemcpy( gpu_sprite, sprite_data, sprite_length, hipMemcpyHostToDevice );
  hipMalloc((void **)&gpu_src, image_size);
  hipMemcpy( gpu_src, data, image_size, hipMemcpyHostToDevice );
  hipMalloc((void **)&gpu_out1, image_size);
  hipMalloc((void **)&gpu_out2, image_size);
  
  // Create events
  hipEventCreate(&start1);
  hipEventCreate(&start2);
  hipEventCreate(&stop1);
  hipEventCreate(&stop2);

  // Create streams
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  // Start Initial Kernels
  hipEventRecord(start1, stream1);
  gpu_img_sprite<<<num_blocks,num_threads,0,stream1>>>(gpu_src,gpu_sprite,
						       sprite_width,sprite_height,
						       gpu_out1,
						       0);
  hipMemcpyAsync( cpu_out1, gpu_out1, image_size, hipMemcpyDeviceToHost, stream1 );
  hipEventRecord(stop1, stream1);
  hipEventRecord(start2, stream2);
  gpu_img_sprite<<<num_blocks,num_threads,0,stream2>>>(gpu_src,gpu_sprite,
						       sprite_width,sprite_height,
						       gpu_out2,
						       1);
  hipMemcpyAsync( cpu_out2, gpu_out2, image_size, hipMemcpyDeviceToHost, stream2 );
  hipEventRecord(stop2, stream2);
  
  // Generate Frames
  for(int i = 0; i < num_images/2; i++)
  {
    hipStreamSynchronize(stream1);

    // Write buffer 0 to disk
    sprintf(out_fn, "%s[%04d].ppm", output_file, 2*i);
    write_image(out_fn, cpu_out1, width, height);

    // Restart stream1 for next iteration (if this isn't the last iteration)
    if (i+1 != num_images/2) {
      hipEventRecord(start1, stream1);
      gpu_img_sprite<<<num_blocks,num_threads,0,stream1>>>(gpu_src,gpu_sprite,
							   sprite_width,sprite_height,
							   gpu_out1,
							   2*(i+1));

      // Copy data to CPU from buffer 0
      hipMemcpyAsync( cpu_out1, gpu_out1, image_size, hipMemcpyDeviceToHost, stream1 );
      hipEventRecord(stop1, stream1);
    }

    hipStreamSynchronize(stream2);
    
    // Write buffer 2 to disk
    sprintf(out_fn, "%s[%04d].ppm", output_file, 2*i+1);
    write_image(out_fn, cpu_out2, width, height);

    // Start next kernel
    if (i+1 != num_images/2) {
      hipEventRecord(start2, stream2);
      gpu_img_sprite<<<num_blocks,num_threads,0,stream2>>>(gpu_src,gpu_sprite,
							   sprite_width,sprite_height,
							   gpu_out2,
							   1+(2*(i+1)));

      // Copy data to CPU from buffer 0
      hipMemcpyAsync( cpu_out2, gpu_out2, image_size, hipMemcpyDeviceToHost, stream2 );
      hipEventRecord(stop2, stream2);
    }
    
    
  }

  // Cleanup
  // free CPU+GPU output buffers
  hipFree(gpu_out1);
  hipFree(gpu_out2);
  hipHostFree(cpu_out1);
  hipHostFree(cpu_out2);
  
  // free Sprite buffers
  hipFree(gpu_sprite);
  hipHostFree(sprite_data);
  
  // Note: main fn will free main image cpu buffer
  hipFree(gpu_src);

  return 0;
}

/** Perform Image Processing functions using the NPP Library NOTE:
 *   While we are loading 3-channel 8-bit RGB images, we will parse as
 *   4-channel images since that data format for NPP matches the
 *   format already used in this file.  The fourth channel (nominally
 *   alpha) is simply ignored with our current import/export
 *   functions.
 */
  int npp_mod_image(uint32_t *data, uint32_t height, uint32_t width)
{
  NppiSize size = {width, height};
  NppStatus status;

  npp::ImageCPU_8u_C4 oHost(width, height);
  memcpy(oHost.data(), data, (width*height*sizeof(uint32_t)) );
  npp::ImageNPP_8u_C4 oDevice(oHost);
  printf("chromakey = %x\n", host_chromakey);
  
  switch(mode) {
  case MODE_AND_MASK:
    status = nppiAndC_8u_C4IR((const Npp8u *)&host_chromakey, oDevice.data(), oDevice.pitch(), size);
    break;
  case MODE_OR_MASK:
    status = nppiOrC_8u_C4IR((const Npp8u *)&host_chromakey, oDevice.data(), oDevice.pitch(), size);
  }

  if (status < 0) {
    printf("ERROR: NPP Operation failed with %i\n");
  }

  oDevice.copyTo(oHost.data(), oHost.pitch());
  memcpy(data, oHost.data(), (width*height*sizeof(uint32_t)) );
  //oDevice.copyTo((Npp8u*)data, oDevice.pitch());
  nppiFree(oDevice.data());
}
  void cuda_img_precheck(uint32_t height, uint32_t width) {
    // Sanity Checks
    if (num_threads == 0) {
      num_threads = width;
    }
    if (num_blocks == 0) {
      //num_blocks = height;
      num_blocks = (width*height)/num_threads;
    }

    if (num_blocks*num_threads > height*width) {
      printf("ERROR: num_blocks&num_threads must be <= image size (%d <= %d)\n", num_blocks*num_threads, height*width);
      throw std::runtime_error("ERROR: Invalid parameters");
    }
    printf("Processing image %s fn of size (%d x %d) with %d threads and %d blocks\n",
           input_file, height, width, num_threads, num_blocks);
}
  
}; // End class GIMP_cuda
  
int main(int argc, char* argv[])
{
#if 1
  // Declare app and initialize
  GIMD_cuda app;

  // Parse Arguments, Execute and run
  return app.run(argc, argv);

#else
  
      case MODE_STEG_DE:
	steg_image_de(num_threads, num_blocks, image, image_size, output_file.c_str(), memMode);
	skip_img_write = 1;
	break;
      case MODE_STEG_EN:
	if (msg.length() == 0) {
	  printf("ERROR: Encryption needs a message to encrypt!\n");
	  break;
	}
	steg_image_en(num_threads, num_blocks, image, image_size, msg.c_str(), memMode);
	break;

#endif
}
